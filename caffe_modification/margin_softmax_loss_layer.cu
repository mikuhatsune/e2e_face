#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#define _USE_MATH_DEFINES
#include <cmath>
#include <vector>

//#include <hip/hip_runtime.h>
//#include "caffe/layer.hpp"
#include "caffe/layers/margin_softmax_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

//#define M_PI       3.14159265358979323846

namespace caffe {

// no need to take 'sqrt'!!!
template <typename Dtype>
void caffe_gpu_norm2(const int n, const Dtype* x, Dtype* out);
template <>
void caffe_gpu_norm2<float>(const int n, const float* x, float* out) {
    CUBLAS_CHECK(hipblasSnrm2(Caffe::cublas_handle(), n, x, 1, out));
}
template <>
void caffe_gpu_norm2<double>(const int n, const double* x, double* out) {
    CUBLAS_CHECK(hipblasDnrm2(Caffe::cublas_handle(), n, x, 1, out));
}

/*template <typename Dtype>
__global__ void sqrt_kernel(const int n, Dtype* y) {
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = sqrt(y[index]);
    }
}*/
template <typename Dtype>
static __global__ void compute_exp_kernel(const int n, const int P, const Dtype *max_f, Dtype* y) {
    CUDA_KERNEL_LOOP(index, n) {
        y[index] = exp(y[index] - max_f[index / P]);
    }
}

__device__ inline void atomic_add(float * address, float val) {
    atomicAdd(address, val);
}
__device__ inline void atomic_add(double * address, double val) {
    unsigned long long int* address_as_ull =
            (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val +
                __longlong_as_double(assumed)));
        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
}

//__constant__ const double cos_k_table[5] = { 1, M_SQRT1_2, 0, -M_SQRT1_2, -1 };

template <typename Dtype>
static __global__ void compute_fyi(const int nthreads, const int P, const int F, const Dtype lambda,
    const Dtype* x, Dtype* x_norm, const Dtype* w, Dtype* w_norm, const Dtype *label,
    Dtype *ip_data, Dtype *phi_data, Dtype *prob_data, Dtype *max_f) {

    const Dtype cos_k_table[5] = { 1, M_SQRT1_2, 0, -M_SQRT1_2, -1 };

    CUDA_KERNEL_LOOP(i, nthreads) {
        const int yi = label[i];

        Dtype fyi = prob_data[i*P + yi];
        ip_data[i] = fyi;

        if (sizeof(Dtype) == sizeof(double)) {
            x_norm[i] = norm(F, (const double*)(x + i*F));
            w_norm[i] = norm(F, (const double*)(w + yi*F));
        }
        else {
            x_norm[i] = normf(F, (const float*)(x + i*F));
            w_norm[i] = normf(F, (const float*)(w + yi*F));
        }

        Dtype xw = x_norm[i] * w_norm[i];
        Dtype cos_th = xw == 0 ? 1 : fyi / xw;

        for (int k = 0; k < 4; ++k) {
            //if (cos(k*M_PI / 4) >= cos_th && cos_th >= cos((k + 1)*M_PI / 4)) {
            if (cos_k_table[k] >= cos_th && cos_th >= cos_k_table[k + 1]) {
                Dtype c2 = cos_th * cos_th;
                Dtype phi = (k & 1 ? (-1) : 1) * (8 * c2 * (c2 - 1) + 1) - 2 * k;
                phi_data[i] = phi;
                fyi = (fyi * lambda + xw * phi) / (1 + lambda);
                prob_data[i*P + yi] = fyi;
                break;
            }
        }

        for (int j = 0; j < P; ++j)
            if (prob_data[i*P + j] > fyi)
                fyi = prob_data[i*P + j];
        max_f[i] = fyi;
    }
}

template <typename Dtype>
static __global__ void compute_loss(const int nthreads, const int P,
    const Dtype *label, Dtype *prob_data, const Dtype *sum_exp, Dtype *loss) {

    CUDA_KERNEL_LOOP(index, nthreads) {
        const int i = index / P;
        const int j = index % P;
        const int yi = label[i];

        prob_data[index] /= sum_exp[i];
        if (j == yi) {
            atomic_add(loss, -log(prob_data[index]));
            //atomic_add(loss, log(sum_exp[i]) - log);
        }
    }
}

template <typename Dtype>
void MarginSoftmaxLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* const x = bottom[0]->gpu_data();
    const Dtype* const label = bottom[1]->gpu_data();
    Dtype* const loss = top[0]->mutable_gpu_data();

    const Dtype* const w = this->blobs_[0]->gpu_data();

    Dtype* const ip_data = ip_.mutable_gpu_data();
    Dtype* const phi_data = ip_.mutable_gpu_diff();
    Dtype* const prob_data = prob_.mutable_gpu_data();

    Dtype* const x_norm = prob_.mutable_gpu_diff(); // N
    Dtype* const w_norm = x_norm + N;               // N!!! not P
    Dtype* max_f = w_norm + N;                      // N
    const Dtype* const ones_P = max_f + N;          // P
    Dtype* const sum_exp = (Dtype*)ones_P + P;      // N

    const Dtype lambda = this->lambda_.get_iter("lambda");
    
    // It turnes out to be TOO SLOW...to switch between modes
    /*//hipblasSetPointerMode(Caffe::cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE);
    // compute |x|
    for (int i = 0; i < N; ++i) {
        caffe_gpu_norm2(F, x + i*F, x_norm + i);
    }
    // compute |w|
    //compute_norms<Dtype> << <CAFFE_GET_BLOCKS(P), CAFFE_CUDA_NUM_THREADS >> >(P, F, w, w_norm);
    for (int j = 0; j < P; j++) {
        caffe_gpu_norm2(F, w + j*F, w_norm + j);
    }
    hipblasSetPointerMode(Caffe::cublas_handle(), HIPBLAS_POINTER_MODE_HOST);*/

    //sqrt_kernel<Dtype> << <CAFFE_GET_BLOCKS(P + N), CAFFE_CUDA_NUM_THREADS >> >(P + N, x_norm);

    // compute inner product: w^T * x
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
        N, P, F, (Dtype)1.,
        x, w, (Dtype)0., prob_data);

    // f_j(j!=yi) = w_j^T * x
    // f_yi = ...
    compute_fyi<Dtype> << <CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS >> >(
        N, P, F, lambda,
        x, x_norm, w, w_norm, label,
        ip_data, phi_data, prob_data, max_f);

    // exp(f_j), max_f is employed to ensure numerical stability
    compute_exp_kernel<Dtype> << <CAFFE_GET_BLOCKS(N*P), CAFFE_CUDA_NUM_THREADS >> >(
        N*P, P, max_f, prob_data);

    // compute the denominator: sum(exp(...))
    caffe_gpu_gemv<Dtype>(CblasNoTrans,
        N, P, (Dtype)1,
        prob_data, ones_P, (Dtype)0, sum_exp);

    // compute and accumalate loss
    caffe_gpu_set(1, Dtype(0), loss);
    compute_loss<Dtype> << <CAFFE_GET_BLOCKS(N*P), CAFFE_CUDA_NUM_THREADS >> >(
        N*P, P, label, prob_data, sum_exp, loss);
    caffe_gpu_scal(1, Dtype(1) / N, loss);
    //top[0]->mutable_cpu_data()[0] = loss / N;*/

}

template <typename Dtype>
static __device__ void atomic_axpy(const int n, const Dtype a, const Dtype* x, Dtype* y) {
    for (int i = 0; i < n; ++i, ++x, ++y) {
        atomic_add(y, a * (*x));
    }
}
template <typename Dtype>
static __device__ void axy(const int n, const Dtype a, const Dtype* x, Dtype* y) {
    for (int i = 0; i < n; ++i, ++x, ++y)
        *y = a * (*x);
}
template <typename Dtype>
static __device__ void axpy(const int n, const Dtype a, const Dtype* x, Dtype* y) {
    for (int i = 0; i < n; ++i, ++x, ++y)
        *y += a * (*x);
}

template <typename Dtype>
static __global__ void backward_yi(const int nthreads, const int P, const int F, const Dtype lambda,
    const Dtype* x, const Dtype* x_norm, const Dtype* w, const Dtype* w_norm, const Dtype* label,
    const Dtype* ip_data, const Dtype* phi_data, Dtype* prob_data,
    Dtype* x_diff, Dtype* w_diff
    ) {

    CUDA_KERNEL_LOOP(i, nthreads) {
        const Dtype * const xi = x + i*F;
        const int yi = label[i];
        Dtype *x_diff_i = x_diff + i*F;
        const Dtype x_norm_i = x_norm[i];
        const Dtype ip = ip_data[i];
        const Dtype phi = phi_data[i];

        const Dtype * const wj = w + yi*F;
        Dtype *w_diff_j = w_diff + yi*F;
        //const Dtype w_norm_j = w_norm[yi];
        const Dtype w_norm_j = w_norm[i];
        const Dtype prob_ij = prob_data[i*P + yi] - 1;

        // avoid mis-computation during the later 'gemm'
        prob_data[i*P + yi] = 0;

        if (w_norm_j == 0) {
            atomic_axpy(F, prob_ij * 4, xi, w_diff_j);
            axy(F, Dtype(0), wj, x_diff_i);    // set to 0
        }
        else if (x_norm_i == 0) {
            axy(F, prob_ij * 4, wj, x_diff_i);
        }
        else {
            Dtype m = ((-1 > phi && phi >= -3) || (-5 > phi && phi >= -7)) * (-8);
            m = m * (2 * ip*ip / (w_norm_j*w_norm_j*x_norm_i*x_norm_i) - 1);

            // d(Li)/d(xi) = -d(f_yi)/d(xi) * ( 1-p(yi|xi,w) ) + \sum_{j!=y_i} w_j * p(j|wi,w)
            //   d( f_yi = |w||x|phi(th) ) / d(x) = 
            axy(F,
                prob_ij * (
                    w_norm_j * phi / x_norm_i
                    - m * 2 * ip*ip / (x_norm_i*x_norm_i*x_norm_i*w_norm_j)
                ) / (1 + lambda),
                xi, x_diff_i);

            // d(Li)/d(w_yi) = -d(f_yi)/d(w_yi) * ( 1-p(yi|xi,w) )
            //   d( f_yi = |w||x|phi(th) ) / d(w) = 
            atomic_axpy(F,
                prob_ij * (
                    x_norm_i * phi / w_norm_j
                    - m * 2 * ip*ip / (w_norm_j*w_norm_j*w_norm_j*x_norm_i)
                ) / (1 + lambda),
                wj, w_diff_j);


            m = prob_ij * (m * 2 * ip / (x_norm_i * w_norm_j) + lambda) / (1 + lambda);

            //atomic_axpy(F, m, wj, x_diff_i);
            axpy(F, m, wj, x_diff_i);

            atomic_axpy(F, m, xi, w_diff_j);
        }
    }
}

template <typename Dtype>
void MarginSoftmaxLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    const Dtype* const x = bottom[0]->gpu_data();
    Dtype* const x_diff = bottom[0]->mutable_gpu_diff();

    const Dtype* const label = bottom[1]->gpu_data();
    const Dtype loss_weight = top[0]->cpu_diff()[0] / N;

    const Dtype* const w = this->blobs_[0]->gpu_data();
    //printf("w_diff[0]=%f\n", this->blobs_[0]->cpu_diff()[0]);
    Dtype* const w_diff = this->blobs_[0]->mutable_gpu_diff();

    const Dtype* const ip_data = ip_.gpu_data();
    const Dtype* const phi_data = ip_.gpu_diff();
    Dtype* const prob_data = prob_.mutable_gpu_data();

    const Dtype* const x_norm = prob_.gpu_diff(); // N
    const Dtype* const w_norm = x_norm + N;       // N!!! not P
    
    const Dtype lambda = this->lambda_.get();

    // compute special cases of j==yi
    backward_yi<Dtype> << <CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS >> >(
        N, P, F, lambda,
        x, x_norm, w, w_norm, label, ip_data, phi_data, prob_data,
        x_diff, w_diff);

    // then, collect gradients from output j!=y_i
    caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
        P, F, N,
        (Dtype)1., prob_data, x,
        (Dtype)1., w_diff);

    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
        N, F, P,
        (Dtype)1., prob_data, w,
        (Dtype)1., x_diff);

    // scale the gradients according to top_loss
    caffe_gpu_scal<Dtype>(P*F, loss_weight, w_diff);
    caffe_gpu_scal<Dtype>(N*F, loss_weight, x_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(MarginSoftmaxLossLayer);

}  // namespace caffe
